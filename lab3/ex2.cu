#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>
#include <iostream>


__global__ void rgb2grayKernel(uchar4* image , int width, int height){
    //Gray = 0.299*R + 0.587*G + 0.114*B
    int index = blockIdx.x * blockDim.x + threadIdx.x ; 
    int stride = blockDim.x * gridDim.x;
    float gray ;
    //process here 
    for ( int i = index ; i < height * width; i += stride) {
        gray = image[i].x * 0.299f + image[i].y * 0.587f + image[i].z * 0.114f;
        image[i] = make_uchar4(gray, gray, gray, image[i].w); // Keep the alpha channel unchanged
    }
}

int main( int argc, char** argv )
{
    //prerequisits for kernel programming 
    int blocksize = 256;
    int width = 1280;
    int height = 720;
    int N = width * height;
    int numBlocks = (N + blocksize - 1 ) / blocksize ;
    
    // create input/output streams
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    if ( !input )
    return 0;

    // capture/display loop
    while (true)
    {
        uchar4* image = NULL;
        int status = 0;

        // can be uchar3, uchar4, float3, float4
        // see videoSource::Status (OK, TIMEOUT, EOS,
        if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT) 
                continue;
                
            break; // EOS
        }
        
        if ( output != NULL )
        {
            //start kernel here before Render
            rgb2grayKernel<<<numBlocks, blocksize>>>(image, width, height);


            //
            output->Render(image, input->GetWidth(), input->GetHeight());
            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);

            if (!output->IsStreaming()) // check if the user quit
            break;
        }
    }
}

// nvcc ex2.cu -o ex2 -ljetson-utils
// ./ex2
//  sudo /usr/local/cuda/bin/nvprof --print-gpu-summary ./ex2
