#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>
#include <iostream>


__global__ void rgb2grayKernel(uchar4* image , int width, int height , uchar4* image_gray){
    //Gray = 0.299*R + 0.587*G + 0.114*B
    int index = blockIdx.x * blockDim.x + threadIdx.x ; 
    int stride = blockDim.x * gridDim.x;
    float gray ;
    //process here 
    for ( int i = index ; i < height * width; i += stride) {
        gray = image[i].x * 0.299f + image[i].y * 0.587f + image[i].z * 0.114f;
        image_gray[i] = make_uchar4(gray, gray, gray, image[i].w); // Keep the alpha channel unchanged
    }
}

int main( int argc, char** argv )
{
    //prerequisits for kernel programming 
    int blocksize = 256;
    int width = 1280;
    int height = 720;
    int N = width * height;
    int numBlocks = (N + blocksize - 1 ) / blocksize ;

    //Allocate memory for pointers 
    //uchar4* h_i_buffer, h_o_buffer; 
    uchar4 *image, *image_gray; 

  
    //allocate device mem
    hipMalloc(&image_gray , N * sizeof(uchar4));
   


    
    // create input/output streams
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output_2 = videoOutput::Create(argc, argv, ARG_POSITION(1));
    if ( !input )
    return 0;

    // capture/display loop
    while (true)
    {
        
        int status = 0;

        // can be uchar3, uchar4, float3, float4
        // see videoSource::Status (OK, TIMEOUT, EOS,
        if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT) 
                continue;
                
            break; // EOS
        }
        
        if ( output != NULL )
        {
            //start kernel here before Render
            rgb2grayKernel<<<numBlocks, blocksize>>>(image, width, height, image_gray);


            //
            output->Render(image, input->GetWidth(), input->GetHeight());
            output_2->Render(image_gray, input->GetWidth(), input->GetHeight());
            
            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);

            if (!output->IsStreaming()) // check if the user quit
            break;
        }
    }
}




// nvcc ex3.cu -o ex3 -ljetson-utils
// ./ex3
//  sudo /usr/local/cuda/bin/nvprof --print-gpu-summary ./ex3
