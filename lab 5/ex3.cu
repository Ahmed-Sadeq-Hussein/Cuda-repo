#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// Error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if (abort) 
            exit(code);
    }
}
//sdata[t] = input[t] + input[t + blocksize]

__global__ void reduceSum(int* input, int* output, int n)
{
    extern __shared__ int partialSum[];
    unsigned int tid = threadIdx.x;
    //unsigned int start = 2*blockIdx.x*blockDim.x;
    // partialSum[tid] = input[start + tid];
    // partialSum[blockDim.x+tid] = input[start + blockDim.x+tid];
    partialSum[tid] = input[tid] + input[tid + blockDim.x];
   
    if (blockDim.x >= 64 && (tid < 32)) {partialSum[tid] += partialSum[tid + 32];}
    if (blockDim.x >=32 && (tid < 16)) {partialSum[tid] += partialSum[tid + 16];}
    if (blockDim.x >= 16 && (tid < 8)) { partialSum[tid] += partialSum[tid + 8];}
    if (blockDim.x >= 8  &&  (tid < 4)) {partialSum[tid] += partialSum[tid + 4];}
    if (blockDim.x >= 4  && (tid < 2)) {partialSum[tid] += partialSum[tid + 2];}
    if (blockDim.x >= 2  && (tid < 1)) {partialSum[tid] += partialSum[tid + 1];}

   
    __syncthreads();
    if (tid == 0)
        output[blockIdx.x] = partialSum[0];
}

int main(void) 
{
    const int numElements = 1 << 24;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (numElements + threadsPerBlock * 2 - 1) / (threadsPerBlock * 2);
    const int smemSize = 2 * threadsPerBlock * sizeof(int);
    int *h_input = (int *)malloc(numElements * sizeof(int));
    int *h_output = (int *)malloc(blocksPerGrid * sizeof(int));

    // Initialize the host input vector
    for (int i = 0; i < numElements; ++i) 
    {
        h_input[i] = rand() % 100;
    }

    int *d_input, *d_output;
    cudaCheckError(hipMalloc((void **)&d_input, numElements * sizeof(int)));
    cudaCheckError(hipMalloc((void **)&d_output, blocksPerGrid * sizeof(int)));
    cudaCheckError(hipMemcpy(d_input, h_input, numElements * sizeof(int), hipMemcpyHostToDevice));

    // Launch the reduction kernel
    reduceSum<<<blocksPerGrid, threadsPerBlock, smemSize>>>(d_input, d_output, numElements);
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipMemcpy(h_output, d_output, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost));

    // Complete the reduction on the CPU
    int totalSum = 0;
    for (int i = 0; i < blocksPerGrid; ++i) 
    {
        totalSum += h_output[i];
    }

    printf("Total Sum (GPU) = %d\n", totalSum);
    int totalSumCPU = 0;

    for (int i = 0; i < numElements; i++) 
    {
        totalSumCPU += h_input[i];
    }

    printf("Total Sum (CPU) = %d\n", totalSumCPU);
    // Free device and host memory
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}