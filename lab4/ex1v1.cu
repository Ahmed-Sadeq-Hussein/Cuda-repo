#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>
#include <iostream>

#define HISTOGRAM_MAX 256
//1280 * 720

__global__ void rgb2grayKernel(uchar4* image , int width, int height , uchar4* image_gray){
    //Gray = 0.299*R + 0.587*G + 0.114*B
    int index = blockIdx.x * blockDim.x + threadIdx.x ; 
    int stride = blockDim.x * gridDim.x;
    float gray ;
    //process here 
    for ( int i = index ; i < 1280 * 720; i += stride) {
        gray = image[i].x * 0.299f + image[i].y * 0.587f + image[i].z * 0.114f;
        image_gray[i] = make_uchar4(gray, gray, gray, image[i].w); // Keep the alpha channel unchanged
    }
}

__global__ void calcHistogramKernel(uchar4* image, int* histo , int width , int height  ) {
    int index = blockIdx.x * blockDim.x + threadIdx.x ; 
    int stride = blockDim.x * gridDim.x;
    int gray ;
    //process here 
    for( int i = index ; i < 1280 * 720 ; i += stride) {
        gray = image[i].x * 0.299f + image[i].y * 0.587f + image[i].z * 0.114f;
        // gray = image[i].x
        histo[gray] += 1; 
    }

}

int main( int argc, char** argv )
{
    //prerequisits for kernel programming 
    int blocksize = 256;
    int width = 1280;
    int height = 720;
    int N = width * height;
    int numBlocks = (N + blocksize - 1 ) / blocksize ;
 ///921600
    ///(((Histogram)))
    int *histo ;
    int temp_calc;

    hipMalloc(&histo , HISTOGRAM_MAX * sizeof(int));
    hipMemset(histo , 0 , 256*sizeof(int)); // to reset all to zero. do after each kernel






    //Allocate memory for pointers 
    //uchar4* h_i_buffer, h_o_buffer 
    uchar4 *image;
    uchar4 *image_gray; 

  
    //allocate device mem
    hipMalloc(&image_gray , N * sizeof(uchar4));
   


    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output_2 = videoOutput::Create(argc, argv, ARG_POSITION(1));
    if ( !input )
    return 0;

    // capture/display loop
    while (true)
    {
        
        int status = 0;

        // can be uchar3, uchar4, float3, float4
        // see videoSource::Status (OK, TIMEOUT, EOS,
        if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT) 
                continue;
                
            break; // EOS
        }
        
        if ( output != NULL )
        {
            //start kernel here before Render
            rgb2grayKernel<<<numBlocks, blocksize>>>(image, width, height, image_gray);
            calcHistogramKernel<<<numBlocks, blocksize>>>(image, histo , width , height);

            for(int i = 0 ; i < 256 ; i++) {
                temp_calc += histo[i];
            }
            printf("%d" , temp_calc);

            
            hipMemset(histo , 0 , 256*sizeof(int));

            //
            output->Render(image, input->GetWidth(), input->GetHeight());
            output_2->Render(image_gray, input->GetWidth(), input->GetHeight());
            
            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);

            if (!output->IsStreaming()) // check if the user quit
            break;
        }
    }
}



// nvcc ex1v1.cu -o ex1v1 -ljetson-utils
// ./ex1v1
//  sudo /usr/local/cuda/bin/nvprof --print-gpu-summary ./ex1v1
