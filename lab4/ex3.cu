#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>
#include <iostream>


//numero uno deus batista


//1280 * 720

__global__ void rgb2grayKernel(uchar4* image , int width, int height , uchar4* image_gray){
    //Gray = 0.299*R + 0.587*G + 0.114*B
    int index = blockIdx.x * blockDim.x + threadIdx.x ; 
    int stride = blockDim.x * gridDim.x;
    float gray ;
    //process here 
    for ( int i = index ; i < 1280 * 720; i += stride) {
        gray = image[i].x * 0.299f + image[i].y * 0.587f + image[i].z * 0.114f;
        image_gray[i] = make_uchar4(gray, gray, gray, image[i].w); // Keep the alpha channel unchanged
    }
}
__global__ void plotHistogramKernel(uchar4* image, int* histogram, int width, int height, int max_freq)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    uchar4 white_pixel = make_uchar4(255, 255, 255, 255);
    uchar4 black_pixel = make_uchar4(0, 0, 0, 255);
    if(index < 256){
        int freq = histogram[index] * 256 / max_freq;
        for(int i = 0 ; i < 256 ; i++){
            int row = height - i - 1;
            if (i <= freq) {
                image[row * width + 2*index] = white_pixel;
                image[row * width + 2*index+1] = white_pixel;
            }
            else {
                image[row * width + 2*index] = black_pixel;
                image[row * width + 2*index+1] = black_pixel;
            }
        }
    }
}

__global__ void makegrayimage (uchar4* image , int width, int height , uchar4* image_gray){
    //Gray = 0.299*R + 0.587*G + 0.114*B
    int index = blockIdx.x * blockDim.x + threadIdx.x ; 
    int stride = blockDim.x * gridDim.x;
    float gray ;
    //process here 
    for ( int i = index ; i < 1280 * 720; i += stride) {
        gray = (256*(i/width))/height;

        image_gray[i] = make_uchar4(gray, gray, gray, image[i].w); // Keep the alpha channel unchanged
    }
}


__global__ void calcHistogramKernel(uchar4* image, int* histo, int width, int height) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

 //intililizationsion of the shared mem 
    __shared__ int histo_local[256];

    if (threadIdx.x < 256) { //needed apparently
        histo_local[threadIdx.x] = 0;
    }
    __syncthreads();

 
    for (int i = index; i < width * height; i += stride) {
        int gray = image[i].x;  // since the image is already grayscale
        atomicAdd(&histo_local[gray], 1);  // atomic add within shared memory
    }

    __syncthreads();

    //updates my histo thats global
    if (threadIdx.x < 256) {
        atomicAdd(&histo[threadIdx.x], histo_local[threadIdx.x]);
    }
}





int main( int argc, char** argv )
{
    //prerequisits for kernel programming 
     int max_feq = 20000; 
    int blocksize = 256;
    int width = 1280;
    int height = 720;
    int N = width * height;
    int numBlocks = (N + blocksize - 1 ) / blocksize ;
 ///921600
    ///(((Histogram)))
    int *histo ;
    int copy_histo[256];
    int temp_val = 0;

    hipMalloc(&histo , 256 * sizeof(int));
    hipMemset(histo , 0 , 256*sizeof(int)); // to reset all to zero. do after each kernel






    //Allocate memory for pointers 
    //uchar4* h_i_buffer, h_o_buffer 
    uchar4 *image;
    uchar4 *image_gray; 

  
    //allocate device mem
    hipMalloc(&image_gray , N * sizeof(uchar4));
   


    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output_2 = videoOutput::Create(argc, argv, ARG_POSITION(1));
    if ( !input )
        return 0;

    // capture/display loop
    while (true)
    {
        
        int status = 0;

        // can be uchar3, uchar4, float3, float4
        // see videoSource::Status (OK, TIMEOUT, EOS,
        if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT) 
                continue;
                
            break; // EOS
        }
        
        if ( output != NULL )
        {
            //start kernel here before Render
            makegrayimage<<<numBlocks, blocksize>>>(image, width, height, image_gray);
            calcHistogramKernel<<<numBlocks, blocksize>>>(image_gray, histo , width , height);
            plotHistogramKernel<<<numBlocks , blocksize>>>(image_gray, histo, width, height, max_feq);
            //printing and calculation
            hipMemcpy(copy_histo, histo, 256 *sizeof(int), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            for(int i = 0 ; i < 256 ; i++) {
                temp_val += copy_histo[i];
            }
            
            
            //reset
            hipMemset(histo , 0 , 256*sizeof(int));
            
            //
            output->Render(image, input->GetWidth(), input->GetHeight());
            output_2->Render(image_gray, input->GetWidth(), input->GetHeight());
            
            // Update status bar
            char str[256];
            sprintf(str, "%d   Camera Viewer (%ux%u) | %0.1f FPS", temp_val,  input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            temp_val = 0;

            if (!output->IsStreaming()) // check if the user quit
            break;
        }
    }
}




// nvcc ex3.cu -o ex3 -ljetson-utils
// ./ex3
//  sudo /usr/local/cuda/bin/nvprof --print-gpu-summary ./ex3
