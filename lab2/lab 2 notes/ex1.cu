
#include <hip/hip_runtime.h>
#include <iostream> 
#include <math.h> 
  
__global__ void multKernel(int n, float* a, float* b, float* c) 
{ 
    for (int i = 0; i < n; i++) { 
        c[i] = a[i] * b[i]; 
    } 
} 
  
int main() { 
    int N = 1<<24; 
    float *h_a, *h_b, *h_c; 
    float *d_a, *d_b, *d_c; 
  
    // Allocate host memory 
    h_a = new float[N]; 
    h_b = new float[N]; 
    h_c = new float[N]; 
  
    // Allocate device memory 
    hipMalloc(&d_a, N * sizeof(float)); 
    hipMalloc(&d_b, N * sizeof(float)); 
    hipMalloc(&d_c, N * sizeof(float)); 
  
    // Initialize host data 
    for (int i = 0; i < N; i++) 
    { 
        h_a[i] = 2.0f; 
        h_b[i] = 3.0f; 
    } 
  
    // Copy data from host to device 
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice); 
  
    // Launch the kernel 
    multKernel<<<1, 1>>>(N, d_a, d_b, d_c); 
  
    // Copy result back to host 
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost); 
  
    // Check result for errors (all values should be 6.0f) 
    float maxError = 0.0f; 
    for (int i = 0; i < N; i++) 
        maxError = fmax(maxError, fabs(h_c[i] - 6.0f)); 
    std::cout << "Max error: " << maxError << std::endl; 
  
    // Clean up 
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c); 
    delete[] h_a; 
    delete[] h_b; 
    delete[] h_c; 
  
    return 0; 
}

/*
/// ASH CODE . Example on how to capture question 1 . 
#include <iostream>
#include <math.h>
#include <cuda_runtime.h>

__global__ void multKernel(int n, float* a, float* b, float* c) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] * b[i];
    }
}

int main() {
    int N = 1<<24; 
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    h_a = new float[N];
    h_b = new float[N];
    h_c = new float[N];

    // Allocate device memory
    cudaMalloc(&d_a, N * sizeof(float));
    cudaMalloc(&d_b, N * sizeof(float));
    cudaMalloc(&d_c, N * sizeof(float));

    // Initialize host data
    for (int i = 0; i < N; i++) {
        h_a[i] = 2.0f;
        h_b[i] = 3.0f;
    }

    // Timing events
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);

    // Copy data from host to device
    cudaEventRecord(start);
    cudaMemcpy(d_a, h_a, N * sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_b, h_b, N * sizeof(float), cudaMemcpyHostToDevice);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    float hostToDeviceTime;
    cudaEventElapsedTime(&hostToDeviceTime, start, stop);

    // Launch the kernel
    cudaEventRecord(start);
    multKernel<<<1, 1>>>(N, d_a, d_b, d_c);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    float kernelTime;
    cudaEventElapsedTime(&kernelTime, start, stop);

    // Copy result back to host
    cudaEventRecord(start);
    cudaMemcpy(h_c, d_c, N * sizeof(float), cudaMemcpyDeviceToHost);
    cudaEventRecord(stop);
    cudaEventSynchronize(stop);
    float deviceToHostTime;
    cudaEventElapsedTime(&deviceToHostTime, start, stop);

    // Output timing results
    std::cout << "Time spent copying data from host to device: " << hostToDeviceTime << " ms" << std::endl;
    std::cout << "Time spent executing the kernel: " << kernelTime << " ms" << std::endl;
    std::cout << "Time spent copying data from device to host: " << deviceToHostTime << " ms" << std::endl;

    // Check result for errors (all values should be 6.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(h_c[i] - 6.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Clean up
    cudaFree(d_a);
    cudaFree(d_b);
    cudaFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    cudaEventDestroy(start);
    cudaEventDestroy(stop);

    return 0;
}



*/